#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>


//
// kernel routine
//

__global__ void my_first_kernel(float *x, float *y)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  //x[tid] = (float) threadIdx.x;
  //printf("%d %f %f", tid, x[tid], y[tid]);
  x[tid] = x[tid] + y[tid];
}


//
// main code
//

int main(int argc, const char **argv)
{
  float *h_x, *h_y, *d_x, *d_y;
  int   nblocks, nthreads, nsize, n;

  // initialise card

  findCudaDevice(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 2;
  //nblocks = 0; //throws error
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  h_x = (float *)malloc(nsize*sizeof(float));
  h_y = (float *)malloc(nsize*sizeof(float));
  for (n=0; n<nsize; n++){
    h_x[n] = n;
    h_y[n] = 100-n;
  }

  checkCudaErrors(hipMalloc((void **)&d_x, nsize*sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_y, nsize*sizeof(float)));

  // copy to device

  checkCudaErrors( hipMemcpy(d_x,h_x,nsize*sizeof(float),
                 hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(d_y,h_y,nsize*sizeof(float),
                  hipMemcpyHostToDevice) );


  // execute kernel

  my_first_kernel<<<nblocks,nthreads>>>(d_x, d_y);
  getLastCudaError("my_first_kernel execution failed\n");

  // copy back results and print them out

  checkCudaErrors( hipMemcpy(h_x,d_x,nsize*sizeof(float),
                 hipMemcpyDeviceToHost) );
  checkCudaErrors( hipMemcpy(h_y,d_y,nsize*sizeof(float),
                  hipMemcpyDeviceToHost) );

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_x[n]);

  // free memory

  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  free(h_x);
  free(h_y);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
